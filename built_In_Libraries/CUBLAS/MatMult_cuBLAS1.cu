/* Matrix Multiplication using cuBLAS */
#include<hipblas.h>
#include<hip/hip_runtime.h>

#include<stdio.h>
#define data_type float
data_type *A, *B, *C;
data_type *DA, *DB, *DC;
void readArray(data_type *X, int rows, int cols)
{
	for(int i=0;i<rows;i++){
		for(int j=0;j<cols;j++){
			X[i*cols+j]=1.0;
		}
	}
}
void printArray(data_type *X, int rows, int cols)
{
	for(int i=0;i<rows;i++){
		for(int j=0;j<cols;j++){
			printf("   %f", X[i*cols+j]);
		}
		printf("\n");
	}
}
int main(int argc, char *argv[])
{
    int n=atoi(argv[1]);
    int size_in_bytes=sizeof(data_type)*n*n;
    A=(float *)malloc(size_in_bytes);
    B=(float *)malloc(size_in_bytes);
    C=(float *)malloc(size_in_bytes);
    readArray(A, n,n);
    readArray(B, n,n);
    hipMalloc((void **)&DA, size_in_bytes);
    hipMalloc((void **)&DB, size_in_bytes);
    hipMalloc((void **)&DC, size_in_bytes);

    //cuBLAS handle
    hipblasHandle_t handle;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipblasCreate(&handle);

    //scaling factors
    data_type alpha=1.0;
    data_type beta=0.0;
    	
    	// Copy from Host to Device
	//cudaMemcpy(DA, A, size_in_bytes, cudaMemcpyHostToDevice);
    	hipblasSetMatrix(n, n, sizeof(data_type), A, n, DA, n);
	//cudaMemcpy(DB, B, size_in_bytes, cudaMemcpyHostToDevice);
    	hipblasSetMatrix(n, n, sizeof(data_type), B, n, DB, n);
	// Calculation C= (alpha * A) * B + (beta * C);
	hipEventRecord(start);
    	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, DA, n, DB, n, &beta, DC, n);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed;
	hipEventElapsedTime(&elapsed, start, stop);
	
	// Copy from Device to Host
	//cudaMemcpy(C, DC, size_in_bytes, cudaMemcpyDeviceToHost);
	hipblasGetMatrix(n, n, sizeof(data_type), DC, n, C, n);
    	hipDeviceSynchronize();
	printf("\n Elapsed Time : %f\n",elapsed);
	printArray(A,n,n);
	printArray(B,n,n);
	printArray(C,n,n);
	return 0;
}
