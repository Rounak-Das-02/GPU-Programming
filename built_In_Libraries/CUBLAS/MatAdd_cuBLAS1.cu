/* Matrix Multiplication using cuBLAS */
#include<hipblas.h>
#include<hip/hip_runtime.h>

#include<stdio.h>
#define data_type float
data_type *A, *B, *C;
data_type *DA, *DB, *DC;
void readArray(data_type *X, int rows, int cols)
{
	for(int i=0;i<rows;i++){
		for(int j=0;j<cols;j++){
			X[i*cols+j]=i;
		}
	}
}
void readMultiplicativeIdentity(data_type *X, int n){
	for(int i=0;i<n;i++){
		X[i*n+i]=1;
	}
}
void printArray(data_type *X, int rows, int cols)
{
	for(int i=0;i<rows;i++){
		for(int j=0;j<cols;j++){
			printf("   %f", X[i*cols+j]);
		}
		printf("\n");
	}
}
int main(int argc, char *argv[])
{
    int n=atoi(argv[1]);
    int size_in_bytes=sizeof(data_type)*n*n;
    A=(data_type *)calloc(size_in_bytes,1);
    B=(data_type *)calloc(size_in_bytes,1);
    C=(data_type *)calloc(size_in_bytes,1);
    readArray(A, n,n);
    readMultiplicativeIdentity(B, n);
    readArray(C,n,n);
    hipMalloc((void **)&DA, size_in_bytes);
    hipMalloc((void **)&DB, size_in_bytes);
    hipMalloc((void **)&DC, size_in_bytes);

    //cuBLAS handle
    hipblasHandle_t handle;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipblasCreate(&handle);

    //scaling factors
    data_type alpha=1.0;
    data_type beta=1.0;
    	
    	// Copy from Host to Device
	//cudaMemcpy(DA, A, size_in_bytes, cudaMemcpyHostToDevice);
    	hipblasSetMatrix(n, n, sizeof(data_type), A, n, DA, n);
	//cudaMemcpy(DB, B, size_in_bytes, cudaMemcpyHostToDevice);
    	hipblasSetMatrix(n, n, sizeof(data_type), B, n, DB, n);
	hipblasSetMatrix(n, n, sizeof(data_type), C, n, DC, n);
	// Calculation C= (alpha * A) * B + (beta * C);
	hipEventRecord(start);
    	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, DA, n, DB, n, &beta, DC, n);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed;
	hipEventElapsedTime(&elapsed, start, stop);
	
	// Copy from Device to Host
	//cudaMemcpy(C, DC, size_in_bytes, cudaMemcpyDeviceToHost);
	hipblasGetMatrix(n, n, sizeof(data_type), DC, n, C, n);
    	hipDeviceSynchronize();
	printf("\n Elapsed Time : %f\n",elapsed);
	// printArray(C, n, n);
	hipFree(DA);
	hipFree(DB);
	hipFree(DC);
	free(A);
	free(B);
	free(C);
	hipblasDestroy(handle);
	return 0;
}
