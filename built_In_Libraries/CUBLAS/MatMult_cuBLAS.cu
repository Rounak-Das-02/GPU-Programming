/* Matrix Multiplication using cuBLAS */
#include<hipblas.h>
#include<hip/hip_runtime.h>

#include<stdio.h>
#define data_type float
data_type *A, *B, *C;
data_type *DA, *DB, *DC;
void readArray(data_type *X, int rows, int cols){
	for(int i=0;i<rows;i++){
		for(int j=0;j<cols;j++){
			X[i*cols+j]=i;
		}
	}
}
void printArray(data_type *X, int rows, int cols){
	for(int i=0;i<rows;i++){
		for(int j=0;j<cols;j++){
			printf("\t %f", X[i*cols+j]);
		}
		printf("\n");
	}
}
int main(int argc, char *argv[]){
    int n=atoi(argv[1]);
    int size_in_bytes=sizeof(data_type)*n*n;
    A=(data_type *)malloc(size_in_bytes);
    B=(data_type *)malloc(size_in_bytes);
    C=(data_type *)malloc(size_in_bytes);
    readArray(A, n,n);
    readArray(B, n,n);
    hipMalloc((void **)&DA, size_in_bytes);
    hipMalloc((void **)&DB, size_in_bytes);
    hipMalloc((void **)&DC, size_in_bytes);

    //cuBLAS handle
    hipblasHandle_t handle;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipblasCreate(&handle);

    //scaling factors
    data_type alpha=1.0;
    data_type beta=1.0;
    	
    	// Copy from Host to Device
	hipMemcpy(DA, A, size_in_bytes, hipMemcpyHostToDevice);
	hipMemcpy(DB, B, size_in_bytes, hipMemcpyHostToDevice);
	//cudaMemcpy(C, DC, size_in_bytes, cudaMemcpyDeviceToHost);
	//printArray(C, n, n);
    	
	// Calculation C= (alpha * A) * B + (beta * C);
	hipEventRecord(start);
    	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, DA, n, DB, n, &beta, DC, n);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed;
	hipEventElapsedTime(&elapsed, start, stop);
	
	// Copy from Device to Host
	hipMemcpy(C, DC, size_in_bytes, hipMemcpyDeviceToHost);
    	hipDeviceSynchronize();
	hipblasDestroy(handle);
	printf("\n Elapsed Time : %f \n",elapsed);
	//printArray(A, n, n);
	//printArray(C, n, n);
	hipFree(DA);
	hipFree(DB);
	hipFree(DC);
	free(A);
	free(B);
	free(C);
	return 0;
}
