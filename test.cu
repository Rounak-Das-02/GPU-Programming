#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

// __global__ void initializeUsingGPU(int *c)
// {
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     c[index] = index + 1;
// }

__global__ void computeUsingGPU(int *res, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (index <= n)
    {
        printf("%d \n", index);
        atomicAdd(res, index);
    }
}

void display(int *ar, int n)
{
    for (int i = 0; i < n; i++)
    {
        printf("%d ", ar[i]);
    }
}

int main()
{

    int n = pow(2, 15);
    n = 100;

    int BLOCK_SIZE = 32;
    int GRID_SIZE = 4;
    printf("%d \n", GRID_SIZE);

    int *ar = (int *)calloc(n, sizeof(int) * n);
    int *arGPU;
    hipMalloc(&arGPU, n * sizeof(int));

    int res = 0;
    int *resGPU;
    hipMalloc(&resGPU, sizeof(int));
    hipMemcpy(resGPU, &res, sizeof(int), hipMemcpyHostToDevice);
    computeUsingGPU<<<GRID_SIZE, BLOCK_SIZE>>>(resGPU, n);
    hipDeviceSynchronize();

    hipMemcpy(&res, resGPU, sizeof(int), hipMemcpyDeviceToHost);
    printf("\n Sum of %d elememts is %d \n", n, res);

    return 0;
}