/* Program to compute a+b */
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void computeUsingGPUs(int *c, int a , int b)
{
	int temp;
	temp=a+b;
	*c=temp;
}

int main(int argc, char *argv[])
{
	int res=0;
	int *resGPU=0;
	hipMalloc((void **)&resGPU, sizeof(int));	
	computeUsingGPUs<<<1,1>>>(resGPU, 10,20);
	hipDeviceSynchronize();
	hipMemcpy(&res, resGPU, sizeof(int), hipMemcpyDeviceToHost); 
	printf("\n Result is %d \n ", res); 
	hipFree(resGPU);
	return 0;
}
