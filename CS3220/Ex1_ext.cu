/* Program to compute a+b */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__ void computeUsingGPUs(int *c, int a, int b)
{
	//*c=*c+a+b;
	int temp = a + b;
	atomicAdd(c, temp);
}
int main(int argc, char *argv[])
{
	int res = 0;
	int *resGPU = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	printf("\n 1. cudaStatus: %d\n", cudaStatus);
	if (cudaStatus != hipSuccess)
	{
		printf("Cuda Set Device Failed %s", hipGetErrorString(cudaStatus));
		return 0;
	}
	cudaStatus = hipMalloc((void **)&resGPU, sizeof(int) * 1000000000);
	printf("\n 2. cudaStatus: %d\n ", cudaStatus);
	if (cudaStatus != 0)
	{
		printf("\n Memory Allocation Error %s", hipGetErrorString(cudaStatus));
		return 0;
	}
	cudaStatus = hipMemcpy(resGPU, &res, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != 0)
	{
		printf("\n hipMemcpy is Failed:%d", cudaStatus);
		return 0;
	}
	computeUsingGPUs<<<1, 4>>>(resGPU, 10, 20);
	// cudaDeviceSynchronize();
	hipMemcpy(&res, resGPU, sizeof(int), hipMemcpyDeviceToHost);
	printf("\n Result is %d \n ", res);
	hipFree(resGPU);

	return 0;
}
