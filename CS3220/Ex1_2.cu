/* Program to compute a+b */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__ void computeUsingGPUs(int *c, int a, int b)
{
	int temp;
	temp = a + b;
	//*c+=temp;
	atomicAdd(c, temp);
}

int main(int argc, char *argv[])
{
	int res = 0;
	int *resGPU = 0;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void **)&resGPU, sizeof(int));
	printf("\n Cuda Status: %d\n ", cudaStatus);
	if (cudaStatus != 0)
	{
		printf("\n Memory Allocation Error");
		return 0;
	}
	hipMemcpy(resGPU, &res, sizeof(int), hipMemcpyHostToDevice); // dest, src, size, direction
	computeUsingGPUs<<<1, 4>>>(resGPU, 10, 20);
	hipDeviceSynchronize();
	hipMemcpy(&res, resGPU, sizeof(int), hipMemcpyDeviceToHost);
	printf("\n Result is %d \n ", res);
	hipFree(resGPU);
	return 0;
}
