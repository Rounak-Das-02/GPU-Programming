/* Program to compute a+b */
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void computeUsingGPUs(int *c, int a , int b)
{
	int temp=a+b;
	atomicAdd(c, temp);
}

int main(int argc, char *argv[])
{
	int res=30;
	int *resGPU;
	hipMalloc((void **)&resGPU, sizeof(int));
	hipMemcpy(resGPU, &res, sizeof(int),hipMemcpyHostToDevice); 
	computeUsingGPUs<<<1,4>>>(resGPU, 10,20);
	hipMemcpy(&res,resGPU, sizeof(int),hipMemcpyDeviceToHost);
	hipDeviceSynchronize(); 
	printf("\n Sum is %d \n", res);
	return 0;
}
