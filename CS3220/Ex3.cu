// Program to compute sum of all elements in an array
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
int *a, sum;	// Host variable
int *da, *dsum; // Device variables
__global__ void computeUsingGPUs(int *res, int *arr, int n)
{
	int temp = 0;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int inc = blockDim.x * gridDim.x;
	for (int i = tid; i < n; i = i + inc)
	{
		temp += arr[i];
	}
	atomicAdd(res, temp);
}
int main(int argc, char *argv[])
{
	int n, numbl, numth;

	n = atoi(argv[1]);	   // Size of the Array
	numbl = atoi(argv[2]); // Number of Blocks
	numth = atoi(argv[3]); // Number of Threads

	hipError_t hipSuccess;
	a = (int *)calloc(n, sizeof(int)); // Allocate Memory on CPU

	hipSuccess = hipMalloc(&da, n * sizeof(int)); // Allocate Memory on the GPU
	if (hipSuccess != 0)
	{
		printf("\n Error1");
		return 0;
	}

	hipMalloc(&dsum, sizeof(int)); // Allocate MEmory on the GPU
	if (hipSuccess != 0)
	{
		printf("\n Error2");
		return 0;
	}

	for (int i = 0; i < n; i++)
	{
		a[i] = i + 1;
	} // Initialization
	// Mem Copy from Host to Device
	hipSuccess = hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);
	if (hipSuccess != 0)
	{
		printf("\n Error3 \n");
		return 0;
	}
	// Mem Copy from Host to Device
	hipSuccess = hipMemcpy(dsum, &sum, sizeof(int), hipMemcpyHostToDevice);
	if (hipSuccess != 0)
	{
		printf("\n Error4 \n");
		return 0;
	}

	// Compute the Sum using a GPU
	computeUsingGPUs<<<numbl, numth>>>(dsum, da, n);

	// Copy back the result from Device to GPU
	hipSuccess = hipMemcpy(&sum, dsum, sizeof(int), hipMemcpyDeviceToHost);
	if (hipSuccess != 0)
	{
		printf("\n Error5 \n");
		return 0;
	}

	// Display the sum
	printf("\n Sum of %d elememts is %d \n", n, sum);
	hipDeviceSynchronize();
	return 0;
}
