// TASK 2

#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <time.h>
#include <hip/hip_runtime.h>



void print(float **h, int N){
	printf("\nFinal Temperatures on Host: \n");
	for (int i = 0; i < N; i+=N/10) {
		for (int j = 0; j < N; j+=N/10) {
			printf("%-.2f\t ", h[i][j]);
		}
		printf("\n");
	}
}

void print_device(float *h, int N){
	printf("\nFinal Temperatures on Device: \n");
	for (int i = 0; i < N; i+=N/10) {
		for (int j = 0; j < N; j+=N/10) {
			printf("%-.2f\t ", h[i*N + j]);
		}
		printf("\n");
	}
}


// Serially computes on the host
__host__ void heat_dist_host_serial(int T, int N, float** h, float** g){
	for (int iterations = 0; iterations < T; iterations++) {
		for (unsigned long long i = 1; i < N - 1; i++) {
			for (unsigned long long j = 1; j < N - 1; j++) {
                if(i == 0.5*N && j >= 0.4*N && j < 0.6*N){
                    continue;
                }
				g[i][j] = 0.25 * (h[i - 1][j] + h[i + 1][j] + h[i][j - 1] + h[i][j + 1]);
			}
		}
		for (unsigned long long i = 1; i < N - 1; i++) {
			for (unsigned long long j = 1; j < N - 1; j++) {
                if(i == 0.5*N && j >= 0.4*N && j < 0.6*N){
                    continue;
                }
				h[i][j] = g[i][j];
			}
		}
	}
}

//Task c
__host__ int test_result(float** h, float* d_h, int N){
    #pragma omp parallel for
    for(int i = 0 ; i < N; i++){
        for(int j = 0 ; j < N; j++){
			
			float num1 = h[i][j];
			float num2 = d_h[i*N+j];
			num1 = round(num1 * 100) / 100; // round num1 to two decimal places
			num2 = round(num2 * 100) / 100; // round num2 to two decimal places

            if(fabs(num1- num2) > 0.1){ // Allowing a small delta. If the numbers are close enough, it is good to go
				printf("%f != %f", num1, num2);
                return 0;
            }
        }
    }
    return 1;
}


__global__ void gpu_heat_dist__arbitrary_grid_and_block_size_kernel(float *d_playground, float *d_temp, int N, int strideLength)
{
	// unsigned int upper = N-1;

	// int col = blockIdx.x*blockDim.x + threadIdx.x;
	// int row = blockIdx.y*blockDim.y + threadIdx.y;

	// Calculate unique thread ID
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	unsigned long long sIndex = threadId * strideLength;
    unsigned long long eIndex = sIndex + strideLength;

	// printf("%d\n", eIndex);

    if (eIndex > N*N)
    {
        eIndex = N*N-1;
    }

	for (unsigned long long i = sIndex; i < eIndex; i++)
    {
        
		if (i < N || i >=  N*(N-1) || i % N == 0 || (i + 1)%N == 0 ){
			continue;
		}

        if ((i >= (int)(0.5*N)*N+(int)(0.4*N) && i < (int)(0.5*N)*N+(int)(0.6*N)) ){
            continue;
        }

        d_temp[i] = 0.25 * (d_playground[(i-1)] + d_playground[(i+1)] + d_playground[(i-N)] + d_playground[(i+N)]);
		// printf("d_temp : %d\n" , d_temp[i]);
    }
}



int main (int argc, char *argv[]) {
	int N, T;
	N = 100;

	printf("Enter the maximum number of iterations:\n");
	scanf("%d", &T);

    // only change these params if variable grid and block size is needed.
	dim3 grid(80, 1);
	dim3 block(32, 32);
	int total = 80*32*32;

	int strideLength = (N*N + total - 1) / total;


	printf("=====================\n");
	printf("Total Points : %d x %d\n", N, N);
	printf("Total Number of Blocks : %d\n", 1);
	printf("Total Number of Threads : %d\n", 1);
	printf("=====================\n");


    //Task a
	printf("Intializing ... \n");
	float **g = new float*[N];
    float **h = new float*[N];
    for (int i = 0; i < N; i++ ) {	//initialize array
        g[i] = new float[N];
        h[i] = new float[N];
		for (int j = 0; j < N; j++) {
			h[i][j] = 0;
			g[i][j] = 0;
		}
	}

	//initialize all walls to temperature of 20C
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			h[0][i] = 20.0;
			h[i][0] = 20.0;
			h[N - 1][i] = 20.0;
			h[i][N - 1] = 20.0;
		}
	}


    // Warm body somewhere !
    for (int i = 0.4*N; i < 0.6*N; i++) {
		h[(int)(0.5*N)][i] = 37.0;
	}




	//define fireplace area
	float fire_start, fire_end;
	fire_start = 0.3 * N;
	fire_end = 0.7 * N;

	//declare temperature of fireplace
	for (int i = fire_start; i < fire_end; i++) {
		h[0][i] = 100.0;
	}

	printf("\n");
	printf("Initial Temperatures: \n");
	for (int i = 0; i < N; i+=N/10) {
		for (int j = 0; j < N; j+=N/10) {
			printf("%-.2f\t", h[i][j]);
		}
		printf("\n");
	}



    clock_t start_, end;
    float cpu_time_used;

    start_ = clock();

    // Serial Computation

	printf("Calculating Temperatures in Host ... \n" );
    heat_dist_host_serial(T, N, h, g);
    end = clock();
    print(h, N);
    cpu_time_used = ((float) (end - start_)) / CLOCKS_PER_SEC;
    printf("Time elapsed in Serial: %f ms\n", cpu_time_used*1000);

    // exit(0);




    // GPU CODE STARTS FROM HERE


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    float *playground = NULL;
    playground = (float *)calloc(N*N, sizeof(float));
    int i;
    for(i = 0; i < N; i++)
        playground[i] = 20;

    for(i = 0; i < N; i++)
        playground[i*N] = 20;

    for(i = 0; i < N; i++)
        playground[i*N + (N-1)] = 20;

    for(i = 0; i < N; i++)
        playground[(N-1)*N + i] = 20;

    
    for (int i = 0.4*N; i < 0.6*N; i++) {
		playground[(int)(0.5*N)*N + i] = 37.0;
	}


    // from 4ft of 10ft
    for(i = (int)N*0.3; i < (int)(N*0.7); i++)
        playground[i] = 100;


    float *d_temp , *d_playground;

    hipMalloc(&d_playground, N*N*sizeof(float));
    hipMalloc(&d_temp, N*N*sizeof(float));

    hipMemcpy(d_temp, playground, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_playground, playground, N*N*sizeof(float), hipMemcpyHostToDevice);


	printf("\nCalculating Temperatures in Device ... \n" );
    hipEventRecord(start);
    for (int k = 0; k < T; k++)
	{
		gpu_heat_dist__arbitrary_grid_and_block_size_kernel<<<grid, block>>>(d_playground, d_temp, N , strideLength);
		hipDeviceSynchronize();
        hipMemcpy(d_playground, d_temp, N*N*sizeof(float),hipMemcpyDeviceToDevice);
	}

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);


    hipMemcpy(playground, d_playground, N*N*sizeof(float), hipMemcpyDeviceToHost);
    print_device(playground, N);
    printf("Execution time: %f ms \n",  milliseconds);

	if(test_result(h, playground, N)){
		printf("PASSED\n");
        printf("SpeedUp : %f\n", cpu_time_used*1000/milliseconds);
	}
	else
	printf("FAILED\n");

	hipFree(d_playground);
	hipFree(d_temp);

}


// 80 blocks, 1024 threads 10 iterations -> 180 seconds or almost 3 mins