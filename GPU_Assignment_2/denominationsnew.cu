
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_DENOMINATIONS 9
#define TARGET_AMOUNT 10

__global__ void countChange(int* ways, int* denominations) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid == 0) {
        ways[0] = 1;
    }
    
    __syncthreads();
    
    for (int i = 0; i < NUM_DENOMINATIONS; i++) {
        if (tid >= denominations[i]) {
            atomicAdd(&ways[tid], ways[tid - denominations[i]]);
        }
        
        __syncthreads();
    }
}

int main() {
    int denominations[NUM_DENOMINATIONS] = {1, 2, 5, 10, 20, 50, 100, 500, 2000};
    int* d_denominations;
    hipMalloc(&d_denominations, NUM_DENOMINATIONS * sizeof(int));
    hipMemcpy(d_denominations, denominations, NUM_DENOMINATIONS * sizeof(int), hipMemcpyHostToDevice);
    
    int ways[TARGET_AMOUNT + 1];
    int* d_ways;
    hipMalloc(&d_ways, (TARGET_AMOUNT + 1) * sizeof(int));
    hipMemset(d_ways, 0, (TARGET_AMOUNT + 1) * sizeof(int));
    
    int threadsPerBlock = 256;
    int numBlocks = (TARGET_AMOUNT + threadsPerBlock - 1) / threadsPerBlock;
    countChange<<<numBlocks, threadsPerBlock>>>(d_ways, d_denominations);
    
    hipMemcpy(ways, d_ways, (TARGET_AMOUNT + 1) * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Total number of distinct ways to make change for 10: %d\n", ways[TARGET_AMOUNT]);
    
    hipFree(d_denominations);
    hipFree(d_ways);
    
    return 0;
}
