
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

int *A, *B;
int sum0, sum1, *dsum0, *dsum1;
int *DA0, *DB0, *DA1, *DB1;
#define NUMTH 128

int * allocMemory(int N, int *X){ X=(int *)calloc(sizeof(int), N);	 return X; }
void readArray(int N, int *X){ 
	        for(int i=0;i<N;i++){
		X[i]=1; 
	}
}
void printArray(int N, int *X){
	for(int i=0;i<N;i++){ printf("%d\t", X[i]); }
}
__global__ void multiplyUsingGPUs(int *GA,int *GB,int *dsum,int size,int strideLength){	
	int sIndex=(blockIdx.x*blockDim.x+threadIdx.x)*strideLength;
	int eIndex=sIndex+strideLength;
	if(eIndex>size){ eIndex=size; }
	int temp=0;
	for(int i=sIndex;i<eIndex;i=i+1){ temp+=GA[i]*GB[i]; }
	atomicAdd(dsum, temp);
}
double elapsedTime(struct timeval t1, struct timeval t2){
	return (double)(t2.tv_sec - t1.tv_sec)+(double) (t2.tv_usec - t1.tv_usec)*1.0e-6;
}
int main(int argc, char *argv[]){
	struct timeval tv1, tv2, tv3, tv4, tv5;
	hipEvent_t start0, stop0, start1, stop1;
	hipEventCreate(&start0);
	hipEventCreate(&stop0);
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	struct timezone tz;
	int size=atoi(argv[1]);
	int numBl=atoi(argv[2]);
	int numTh=atoi(argv[3]);
	int N=size;
	gettimeofday(&tv1,&tz);
	hipHostAlloc((void **)&A, N*sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&B, N*sizeof(int), hipHostMallocDefault);
	readArray(N,A);
	readArray(N,B);
	int total=numBl*numTh;
	int strideLength=(N/2+total-1)/total;
	gettimeofday(&tv2, &tz);
	hipMalloc((void **)&DA0, sizeof(int)*N/2);
	hipMalloc((void **)&DB0, sizeof(int)*N/2);
	hipMalloc((void **)&dsum0, sizeof(int));
	hipMalloc((void **)&DA1, sizeof(int)*N/2);
	hipMalloc((void **)&DB1, sizeof(int)*N/2);
	hipMalloc((void **)&dsum1, sizeof(int));
	gettimeofday(&tv3, &tz);
	hipMemcpyAsync(DA0, A, sizeof(int)*N/2, hipMemcpyHostToDevice,stream0);
        hipMemcpyAsync(DB0, B, sizeof(int)*N/2, hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(dsum0, &sum0, sizeof(int), hipMemcpyHostToDevice,stream0);
	
	hipEventRecord(start0, stream0);	
        multiplyUsingGPUs<<<numBl, numTh, 0, stream0>>>(DA0, DB0, dsum0, N/2, strideLength);
	hipEventRecord(stop0, stream0);
	//cudaDeviceSynchronize();

	hipMemcpyAsync(DA1, A+N/2, sizeof(int)*N/2, hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(DB1, B+N/2, sizeof(int)*N/2, hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(dsum1, &sum1, sizeof(int), hipMemcpyHostToDevice, stream1);
	hipEventRecord(start1, stream1);
	multiplyUsingGPUs<<<numBl, numTh, 0, stream1>>>(DA1, DB1, dsum1, N/2, strideLength);
	hipEventRecord(stop1, stream1);
	int totalSum=0;
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	hipMemcpyAsync(&sum0, dsum0, sizeof(int), hipMemcpyDeviceToHost, stream0);
	hipMemcpyAsync(&sum1, dsum1, sizeof(int), hipMemcpyDeviceToHost, stream1);
	totalSum=sum0+sum1;
	gettimeofday(&tv5, &tz);
	printf("Dot Product is %d", totalSum);
	printf("\n Total Execution Time      : %lf\n",elapsedTime(tv1,tv5));
        printf("\n CPU Memory Alloc and Initial Time : %lf\n",elapsedTime(tv1,tv2));
        printf("\n GPU Memory Alloc and Initial Time : %lf\n",elapsedTime(tv2,tv3));	
	printf("\n Execution Time: %lf\n", elapsedTime(tv3,tv5));
	hipEventSynchronize(stop0);
	hipEventSynchronize(stop1);
	float elapsedTime0,elapsedTime1;
	hipEventElapsedTime(&elapsedTime0, start0, stop0);
	hipEventElapsedTime(&elapsedTime1, start1, stop1);
	printf("\n  Stream0 Time: %lf\n Stream1 Time: %lf\n", elapsedTime0/1000, elapsedTime1/1000);
	if(totalSum==N) { printf("PASS\n"); }
	else{ printf("FAIL\n"); }
	hipHostFree(A);
	hipHostFree(B);
	hipFree(DA0);
	hipFree(DB0);
	hipFree(DA1);
	hipFree(DB1);
	return 0;
}
