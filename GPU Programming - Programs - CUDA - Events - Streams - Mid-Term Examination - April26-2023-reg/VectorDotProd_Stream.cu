
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

int *A, *B;
int sum0, *dsum0;
int *DA0, *DB0;

int * allocMemory(int N, int *X){ X=(int *)calloc(sizeof(int), N);	 return X; }
void readArray(int N, int *X){ 
	        for(int i=0;i<N;i++){
		X[i]=1; 
	}
}
void printArray(int N, int *X){
	for(int i=0;i<N;i++){ printf("%d\t", X[i]); }
}
__global__ void multiplyUsingGPUs(int *GA,int *GB,int *dsum,int size,int strideLength){	
	int sIndex=(blockIdx.x*blockDim.x+threadIdx.x)*strideLength;
	int eIndex=sIndex+strideLength;
	if(eIndex>size){ eIndex=size; }
	int temp=0;
	for(int i=sIndex;i<eIndex;i=i+1){ temp+=GA[i]*GB[i]; }
	atomicAdd(dsum, temp);
}
double elapsedTime(struct timeval t1, struct timeval t2){
	return (double)(t2.tv_sec - t1.tv_sec)+(double) (t2.tv_usec - t1.tv_usec)*1.0e-6;
}
int main(int argc, char *argv[]){
	struct timeval tv1, tv2, tv3, tv4, tv5;
	hipEvent_t start0, stop0;
	hipEventCreate(&start0);
	hipEventCreate(&stop0);
	hipStream_t stream0;
	hipStreamCreate(&stream0);
	struct timezone tz;
	int size=atoi(argv[1]);
	int numBl=atoi(argv[2]);
	int numTh=atoi(argv[3]);
	int N=size;
	gettimeofday(&tv1,&tz);
	hipHostAlloc((void **)&A, N*sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&B, N*sizeof(int), hipHostMallocDefault);
	readArray(N,A);
	readArray(N,B);
	int total=numBl*numTh;
	int strideLength=(N+total-1)/total;
	gettimeofday(&tv2, &tz);
	
	hipMalloc((void **)&DA0, sizeof(int)*N);
	hipMalloc((void **)&DB0, sizeof(int)*N);
	hipMalloc((void **)&dsum0, sizeof(int));
	
	gettimeofday(&tv3, &tz);
	hipMemcpyAsync(DA0, A, sizeof(int)*N, hipMemcpyHostToDevice,stream0);
        hipMemcpyAsync(DB0, B, sizeof(int)*N, hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(dsum0, &sum0, sizeof(int), hipMemcpyHostToDevice,stream0);
	
	hipEventRecord(start0, stream0);	
        multiplyUsingGPUs<<<numBl, numTh, 0, stream0>>>(DA0, DB0, dsum0, N, strideLength);
	hipEventRecord(stop0, stream0);
	//cudaDeviceSynchronize();

	int totalSum=0;
	hipStreamSynchronize(stream0);
	hipMemcpyAsync(&sum0, dsum0, sizeof(int), hipMemcpyDeviceToHost, stream0);
	totalSum=sum0;
	gettimeofday(&tv5, &tz);
	printf("Dot Product is %d", totalSum);
	printf("\n Total Execution Time      : %lf\n",elapsedTime(tv1,tv5));
        printf("\n CPU Memory Alloc and Initial Time : %lf\n",elapsedTime(tv1,tv2));
        printf("\n GPU Memory Alloc and Initial Time : %lf\n",elapsedTime(tv2,tv3));	
	printf("\n Execution Time: %lf\n", elapsedTime(tv3,tv5));
	hipEventSynchronize(stop0);
	float elapsedTime0;
	hipEventElapsedTime(&elapsedTime0, start0, stop0);
	printf("\n  Stream0 Time: %lf\n", elapsedTime0/1000);
	if(totalSum==N) { printf("PASS\n"); }
	else{ printf("FAIL\n"); }
	hipHostFree(A);
	hipHostFree(B);
	hipFree(DA0);
	hipFree(DB0);

	return 0;
}
