
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

int *A, *B;
int sum0, sum1, sum2, sum3, *dsum0, *dsum1, *dsum2, *dsum3;
int *DA0, *DB0, *DA1, *DB1, *DA2, *DB2, *DA3, *DB3;
#define NUMTH 128

int * allocMemory(int N, int *X){ X=(int *)calloc(sizeof(int), N);	 return X; }
void readArray(int N, int *X){ 
	        for(int i=0;i<N;i++){
		X[i]=1; 
	}
}
void printArray(int N, int *X){
	for(int i=0;i<N;i++){ printf("%d\t", X[i]); }
}




__global__ void multiplyUsingGPUs(int *GA,int *GB,int *dsum,int size,int strideLength){	
	int sIndex=(blockIdx.x*blockDim.x+threadIdx.x)*strideLength;
	int eIndex=sIndex+strideLength;
	if(eIndex>size){ eIndex=size; }
	int temp=0;
	for(int i=sIndex;i<eIndex;i=i+1){ temp+=GA[i]*GB[i]; }
	atomicAdd(dsum, temp);
	//printf("dsum=%d\n", *dsum);
}






double elapsedTime(struct timeval t1, struct timeval t2){
	return (double)(t2.tv_sec - t1.tv_sec)+(double) (t2.tv_usec - t1.tv_usec)*1.0e-6;
}
int main(int argc, char *argv[]){
	struct timeval tv1, tv2, tv3, tv4, tv5;
	hipEvent_t start0, stop0, start1, stop1, start2, stop2, start3, stop3;
	hipEventCreate(&start0);
	hipEventCreate(&stop0);
	
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	
	hipEventCreate(&start2);
	hipEventCreate(&stop2);

	hipEventCreate(&start3);
	hipEventCreate(&stop3);

	hipStream_t stream0, stream1, stream2, stream3;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);

	struct timezone tz;
	int size=atoi(argv[1]);
	int numBl=atoi(argv[2]);
	int numTh=atoi(argv[3]);
	int N=size;
	gettimeofday(&tv1,&tz);
	hipHostAlloc((void **)&A, N*sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&B, N*sizeof(int), hipHostMallocDefault);
	readArray(N,A);
	readArray(N,B);
	int total=numBl*numTh;
	int strideLength=(N/2+total-1)/total;
	gettimeofday(&tv2, &tz);
	hipMalloc((void **)&DA0, sizeof(int)*N/4);
	hipMalloc((void **)&DB0, sizeof(int)*N/4);
	hipMalloc((void **)&dsum0, sizeof(int));
	hipMalloc((void **)&DA1, sizeof(int)*N/4);
	hipMalloc((void **)&DB1, sizeof(int)*N/4);
	hipMalloc((void **)&dsum1, sizeof(int));
	
	hipMalloc((void **)&DA2, sizeof(int)*N/4);
	hipMalloc((void **)&DB2, sizeof(int)*N/4);
	hipMalloc((void **)&dsum2, sizeof(int));
	
	hipMalloc((void **)&DA3, sizeof(int)*N/4);
	hipMalloc((void **)&DB3, sizeof(int)*N/4);
	hipMalloc((void **)&dsum3, sizeof(int));
	
	gettimeofday(&tv3, &tz);
	hipMemcpyAsync(DA0, A, sizeof(int)*N/4, hipMemcpyHostToDevice,stream0);
        hipMemcpyAsync(DB0, B, sizeof(int)*N/4, hipMemcpyHostToDevice,stream0);
	hipMemcpyAsync(dsum0, &sum0, sizeof(int), hipMemcpyHostToDevice,stream0);
	
	hipEventRecord(start0, stream0);
        multiplyUsingGPUs<<<numBl, numTh, 0, stream0>>>(DA0, DB0, dsum0, N/4, strideLength);
	hipEventRecord(stop0, stream0);

	hipMemcpyAsync(DA1, A+N/4, sizeof(int)*N/4, hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(DB1, B+N/4, sizeof(int)*N/4, hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(dsum1, &sum1, sizeof(int), hipMemcpyHostToDevice, stream1);
	hipEventRecord(start1, stream1);
	multiplyUsingGPUs<<<numBl, numTh, 0, stream1>>>(DA1, DB1, dsum1, N/4, strideLength);
	hipEventRecord(stop1, stream1);


	hipMemcpyAsync(DA2, A+N/2, sizeof(int)*N/4, hipMemcpyHostToDevice,stream2);
	hipMemcpyAsync(DB2, B+N/2, sizeof(int)*N/4, hipMemcpyHostToDevice,stream2);
	hipMemcpyAsync(dsum2, &sum2, sizeof(int), hipMemcpyHostToDevice,stream2);

	hipEventRecord(start2, stream2);
	multiplyUsingGPUs<<<numBl, numTh, 0, stream2>>>(DA2, DB2, dsum2, N/4, strideLength);
	hipEventRecord(stop2, stream2);

	hipMemcpyAsync(DA3, A+3*N/4, sizeof(int)*N/4, hipMemcpyHostToDevice, stream3);
	hipMemcpyAsync(DB3, B+3*N/4, sizeof(int)*N/4, hipMemcpyHostToDevice, stream3);
	hipMemcpyAsync(dsum3, &sum3, sizeof(int), hipMemcpyHostToDevice, stream3);
	hipEventRecord(start3, stream3);
	multiplyUsingGPUs<<<numBl, numTh, 0, stream3>>>(DA3, DB3, dsum3, N/4, strideLength);
	hipEventRecord(stop3, stream3);

	int totalSum=0;
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	hipStreamSynchronize(stream3);

	hipMemcpyAsync(&sum0, dsum0, sizeof(int), hipMemcpyDeviceToHost, stream0);
	hipMemcpyAsync(&sum1, dsum1, sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(&sum2, dsum2, sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(&sum3, dsum3, sizeof(int), hipMemcpyDeviceToHost, stream3);

	totalSum=sum0+sum1+sum2+sum3;
	gettimeofday(&tv5, &tz);
	printf("Dot Product is %d", totalSum);
	printf("\n Total Execution Time      : %lf\n",elapsedTime(tv1,tv5));
        printf("\n CPU Memory Alloc and Initial Time : %lf\n",elapsedTime(tv1,tv2));
        printf("\n GPU Memory Alloc and Initial Time : %lf\n",elapsedTime(tv2,tv3));	
	printf("\n Execution Time: %lf\n", elapsedTime(tv3,tv5));
	hipEventSynchronize(stop0);
	hipEventSynchronize(stop1);
	hipEventSynchronize(stop2);
	hipEventSynchronize(stop3);
	float elapsedTime0,elapsedTime1, elapsedTime2, elapsedTime3;
	hipEventElapsedTime(&elapsedTime0, start0, stop0);
	hipEventElapsedTime(&elapsedTime1, start1, stop1);
	hipEventElapsedTime(&elapsedTime2, start2, stop2);
	hipEventElapsedTime(&elapsedTime3, start3, stop3);

	printf("\n Stream0 Time: %lf\n Stream1 Time: %lf Stream2 Time:%lf Stream3 Time: %lf\n", elapsedTime0/1000, elapsedTime1/1000, elapsedTime2/1000, elapsedTime3/1000);
	if(totalSum==N) { printf("PASS\n"); }
	else{ printf("FAIL\n"); }
	hipHostFree(A);
	hipHostFree(B);
	hipFree(DA0);
	hipFree(DB0);
	hipFree(DA1);
	hipFree(DB1);
	hipFree(DA2);
	hipFree(DB2);
	hipFree(DA3);
	hipFree(DB3);
	return 0;
}
